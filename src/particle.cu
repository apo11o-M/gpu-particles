#include "hip/hip_runtime.h"
#include "particle.hpp"

// Cuda stuff are up here
// ============================================================================

// Macro to check for CUDA API call errors
void cudaErrorCheck(hipError_t res, const char *func, const char *file,
                    int line, bool abort = true) {
    if (res == hipSuccess) return;
    fprintf(stderr,
            "CUDA ERROR!\n\tFunction: %s\n\tLine: %d\n\tFile: %s\n\tError Name:"
            "%s\n\tError Description: %s\n\t",
            func, line, file, hipGetErrorName(res), hipGetErrorString(res));
    if (abort) exit(res);
}
#define cudaAssert(func) { cudaErrorCheck((func), #func, __FILE__, __LINE__); }

__constant__ unsigned int d_maxParticleCount;
__constant__ float d_radius;
__constant__ float d_mass;
__constant__ float d_restitution;
__constant__ float d_dampingFactor;
__constant__ float d_dampingFactorRate;
__constant__ unsigned int d_borderLeft, d_borderRight, d_borderTop, d_borderBottom;
__constant__ unsigned int d_cellXCount, d_cellYCount;
__constant__ float d_maxSuctionRange, d_suctionForce;
__constant__ float d_maxRepelRange, d_repelForce;

// ============================================================================

Particles::Particles(const SimulationConfig& config)
    : r(config.maxParticleCount, 255),
      g(config.maxParticleCount, 255),
      b(config.maxParticleCount, 255),
      position(config.maxParticleCount, Vec2<float>(0, 0)),
      velocity(config.maxParticleCount, Vec2<float>(0, 0)),
      vertices(sf::Quads, config.maxParticleCount * 4),
      renderingThreads(thread::hardware_concurrency() / 2),
      chunkSize(config.maxParticleCount / renderingThreads) {
    
    currActiveIndex = -1;
    maxParticleCount = config.maxParticleCount;
    
    radius = config.particleRadius;
    mass = config.particleMass;
    restitution = config.restitutionCoefficient;
    dampingFactor = config.velocityDampingFactor;
    dampingFactorRate = config.velocityDampingFactorRate;
    maxSuctionRange = config.maxSuctionRange;
    suctionForce = config.suctionForce;
    maxRepelRange = config.maxRepelRange;
    repelForce = config.repelForce;

    borderLeft = config.borderLeft;
    borderRight = config.borderRight;
    borderTop = config.borderTop;
    borderBottom = config.borderBottom;

    cellSize = config.particleRadius * 2.5;
    cellXCount = (borderRight - borderLeft) / cellSize;
    cellYCount = (borderBottom - borderTop) / cellSize;

    cout << "Maximum particle count: " << maxParticleCount << endl;
    cout << "Borders: " << borderLeft << ", " << borderRight << ", " << borderTop << ", " << borderBottom << endl;
    cout << "Grid Count, X: " << cellXCount << ", Y: " << cellYCount << endl;

    for (size_t i = 0; i < maxParticleCount; i++) {
        r[i] = rand() % 255;
        g[i] = rand() % 255;
        b[i] = rand() % 255;
    }

    mouseXPos = 0;
    mouseYPos = 0;
    spawn = FALSE;
    succ = FALSE;
    repel = FALSE;

    if (texture.loadFromFile("assets/circle.png")) {
        cout << "Texture loaded successfully" << endl;
    } else {
        cerr << "Failed to load texture, abort" << endl;
        exit(-1);
    }
    texture.setSmooth(true);
    texture.generateMipmap();

    spawnCount = config.spawnCount;

    // ========================================================================
    // Cuda stuff are down here

    // initialize the constant memory in the device
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_maxParticleCount), &maxParticleCount, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_radius), &radius, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_mass), &mass, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_restitution), &restitution, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_dampingFactor), &dampingFactor, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_dampingFactorRate), &dampingFactorRate, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_maxSuctionRange), &maxSuctionRange, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_suctionForce), &suctionForce, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_maxRepelRange), &maxRepelRange, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_repelForce), &repelForce, sizeof(float)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_borderLeft), &borderLeft, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_borderRight), &borderRight, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_borderTop), &borderTop, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_borderBottom), &borderBottom, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_cellXCount), &cellXCount, sizeof(unsigned int)));
    cudaAssert(hipMemcpyToSymbol(HIP_SYMBOL(d_cellYCount), &cellYCount, sizeof(unsigned int)));

    // allocate memory for the particles in the device
    cudaAssert(hipMalloc(&d_position, maxParticleCount * sizeof(Vec2<float>)));
    cudaAssert(hipMalloc(&d_velocity, maxParticleCount * sizeof(Vec2<float>)));
    cudaAssert(hipMemcpy(d_position, position.data(), maxParticleCount * sizeof(Vec2<float>), hipMemcpyHostToDevice));
    cudaAssert(hipMemcpy(d_velocity, velocity.data(), maxParticleCount * sizeof(Vec2<float>), hipMemcpyHostToDevice));

    // from Matthias Müller, Ten Minute Physics, having the table size be twice 
    // the size of the particle count is a good rule of thumb most of the time
    cudaAssert(hipMalloc(&d_spatialHashTable, (maxParticleCount * 2 + 1) * sizeof(int)));
    cudaAssert(hipMemset(d_spatialHashTable, 0, (maxParticleCount * 2 + 1) * sizeof(int)));
    cudaAssert(hipMalloc(&d_particleIndices, maxParticleCount * sizeof(int)));

    hipDeviceProp_t properties;
    cudaAssert(hipGetDeviceProperties(&properties, 0));
    gpu_maxBlockCount = properties.maxGridSize[0];
    gpu_maxThreadCount = properties.maxThreadsPerBlock;
    if (gpu_maxBlockCount < maxParticleCount) {
        cout << "The number of particles is too large for the device, abort\n"
             << "GPU max grid size: " << gpu_maxBlockCount
             << ", which is too low for " << maxParticleCount << " particles."
             << endl;
        abort();
    }
}

Particles::~Particles() {
    cudaAssert(hipFree(d_position));
    cudaAssert(hipFree(d_velocity));
    cudaAssert(hipFree(d_spatialHashTable));
    cudaAssert(hipFree(d_particleIndices));
}

void Particles::spawnParticles(unsigned int x, unsigned int y, BOOL shouldSpawn) {
    if (shouldSpawn) {
        mouseXPos = x;
        mouseYPos = y;
        spawn = TRUE;
    } else {
        spawn = FALSE;
    }
}

void Particles::succParticles(unsigned int xPos, unsigned int yPos, BOOL shouldSucc) {
    if (shouldSucc) {
        mouseXPos = xPos;
        mouseYPos = yPos;
        succ = TRUE;
    } else {
        succ = FALSE;
    }
}

void Particles::repelParticles(unsigned int xPos, unsigned int yPos, BOOL shouldRepel) {
    if (shouldRepel) {
        mouseXPos = xPos;
        mouseYPos = yPos;
        repel = TRUE;
    } else {
        repel = FALSE;
    }
}

void Particles::render(sf::RenderWindow &window, float deltaTime) {
    BS::multi_future<void> future = threadpool.submit_loop(
        0, currActiveIndex,
        [this, deltaTime](int i) {
            const float textureSize = 1024.0f;
            if (i >= currActiveIndex) return;

            // interpolating the position to achieve smoother movement
            float x = position[i].x - radius + velocity[i].x * deltaTime;
            float y = position[i].y - radius + velocity[i].y * deltaTime;

            vertices[i * 4 + 0].position = sf::Vector2f(x - radius, y - radius);
            vertices[i * 4 + 1].position = sf::Vector2f(x + radius, y - radius);
            vertices[i * 4 + 2].position = sf::Vector2f(x + radius, y + radius);
            vertices[i * 4 + 3].position = sf::Vector2f(x - radius, y + radius);

            vertices[i * 4 + 0].texCoords = sf::Vector2f(0.0f, 0.0f);
            vertices[i * 4 + 1].texCoords = sf::Vector2f(textureSize, 0.0f);
            vertices[i * 4 + 2].texCoords = sf::Vector2f(textureSize, textureSize);
            vertices[i * 4 + 3].texCoords = sf::Vector2f(0.0f, textureSize);

            vertices[i * 4 + 0].color = sf::Color(r[i], g[i], b[i]);
            vertices[i * 4 + 1].color = sf::Color(r[i], g[i], b[i]);
            vertices[i * 4 + 2].color = sf::Color(r[i], g[i], b[i]);
            vertices[i * 4 + 3].color = sf::Color(r[i], g[i], b[i]);   
        }
    );
    future.wait();
    window.draw(vertices, &texture);
}

// clamp the value between min and max
__device__ float clamp(float val, float min, float max) {
    return fmaxf(min, fminf(max, val));
}

// linear iterpolation
__device__ float lerp(const float n1, const float n2, const float time) {
	return n1 + time * (n2 - n1);
}

__global__ void spawnParticleKernel(Vec2<float> *position, Vec2<float> *velocity, 
                                    Vec2<float> mousePosition, int currIndex) {
    // give each spawned particles some offset to avoid overlapping, not perfect
    // but it's good enough
    position[currIndex + threadIdx.x].x = d_borderLeft + 100;
    position[currIndex + threadIdx.x].y = d_borderTop + 100 + threadIdx.x * d_radius * 2;
    velocity[currIndex + threadIdx.x].x = 800;
    velocity[currIndex + threadIdx.x].y = 200;
    // position[currIndex + threadIdx.x].x = mousePosition.x + threadIdx.x;
    // position[currIndex + threadIdx.x].y = mousePosition.y + threadIdx.x;
    // velocity[currIndex + threadIdx.x].x = 0;
    // velocity[currIndex + threadIdx.x].y = 0;
}

// Succ the particles to where the mouse is clicked
__global__ void succParticlesKernel(Vec2<float> *position, Vec2<float> *velocity, 
                                    const int currActiveIndex, const Vec2<float> mousePos) {
    int i = blockIdx.x;
    if (i > currActiveIndex) return;

    // don't succ the particle if it's too far away
    Vec2<float> delta = mousePos - position[i];
    if (delta.lengthSq() > powf(d_maxSuctionRange, 2)) return;

    Vec2<float> deltaNorm = delta.normalized();
    velocity[i] += deltaNorm * lerp(0.0f, d_suctionForce, delta.length() / d_maxSuctionRange);
}

// repel the particles from where the mouse is clicked
__global__ void repelParticlesKernel(Vec2<float> *position, Vec2<float> *velocity,
                                     const int currActiveIndex, const Vec2<float> mousePos) {
    int i = blockIdx.x;
    if (i > currActiveIndex) return;

    // don't repel the particle if it's too far away
    Vec2<float> delta = mousePos - position[i];
    if (delta.lengthSq() > powf(d_maxRepelRange, 2)) return;

    Vec2<float> deltaNorm = delta.normalized();
    velocity[i] -= deltaNorm * lerp(0.0f, d_repelForce, delta.length() / d_maxRepelRange);
}

// spatial hash function, using two large prime numbers to avoid collisions
__device__ int spatialHash(int cellX, int cellY) {
    int res = (cellX * 92837111) ^ (cellY * 689287499);
    return res % (d_maxParticleCount * 2);
}

__global__ void createSpatialHashTable(Vec2<float> *position, int *cellIndices, int *particleIndices, float cellSize) {
    // very important to initialize them to be 0, as this cellIndices hashtable 
    // is used in the previous physics frame update.
    for (int i = threadIdx.x; i < d_maxParticleCount * 2 + 1; i += blockDim.x) {
        cellIndices[i] = 0;
    }
    __syncthreads();

    // count the number of particles in each cell
    for (int i = threadIdx.x; i < d_maxParticleCount; i += blockDim.x) {
        int cellX = (int)((position[i].x - d_borderLeft) / cellSize);
        int cellY = (int)((position[i].y - d_borderTop) / cellSize);
        int hashIndex = spatialHash(cellX, cellY);
        atomicAdd(&cellIndices[hashIndex], 1);
    }
    __syncthreads();

    // prefix sum
    if (threadIdx.x == 0) {
        for (int i = 1; i < d_maxParticleCount * 2 + 1; i++) {
            cellIndices[i] += cellIndices[i - 1];
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < d_maxParticleCount; i += blockDim.x) {
        int cellX = (int)((position[i].x - d_borderLeft) / cellSize);
        int cellY = (int)((position[i].y - d_borderTop) / cellSize);
        int hashIndex = spatialHash(cellX, cellY);
        int index = atomicSub(&cellIndices[hashIndex], 1) - 1;
        particleIndices[index] = i;
    }
}

// The approach here is to have each block process one particle's collision
// against all other ones. Meaning n particles = n blocks. This also means each
// block will have n number of threads.
// There is a better way to approach this, which would require more complex
// index mapping but allow more efficient gpu utilization. 
__global__ void updateKernel(Vec2<float> *position, Vec2<float> *velocity,
                             const int currActiveIndex, 
                             const int *spatialHashtable, const int *particleIndices,
                             const float deltaTime, const float gravity,
                             const float cellSize) {
    // represents the sum of all posDelta and velDelta of one particle
    __shared__ float posDeltaX;
    __shared__ float posDeltaY;
    __shared__ float velDeltaX;
    __shared__ float velDeltaY;

    // only the first thread in the block will update the shared memory
    if (threadIdx.x == 0) {
        posDeltaX = 0;
        posDeltaY = 0;
        velDeltaX = 0;
        velDeltaY = 0;
    }
    __syncthreads();

    // represents the delta of the collided particle
    Vec2<float> posDelta = Vec2<float>(0.0f, 0.0f);
    Vec2<float> velDelta = Vec2<float>(0.0f, 0.0f);

    int base = blockIdx.x;
    int i = particleIndices[base];
    if (i > currActiveIndex) return;
    int cellXPos = (int)((position[i].x - d_borderLeft) / cellSize);
    int cellYPos = (int)((position[i].y - d_borderTop) / cellSize);

    // check for collisions within cells and neighboring cells
    for (int offsetY = -1; offsetY <= 1; offsetY++) {
        for (int offsetX = -1; offsetX <= 1; offsetX++) {
            int neighborX = cellXPos + offsetX;
            int neighborY = cellYPos + offsetY;
            if (neighborX < 0 || neighborX >= d_cellXCount 
                || neighborY < 0 || neighborY >= d_cellYCount) continue;

            int hash = spatialHash(neighborX, neighborY);
            int startIndex = spatialHashtable[hash];
            for (int k = threadIdx.x + startIndex; 
                    k < d_maxParticleCount && k < spatialHashtable[hash + 1]; 
                    k += blockDim.x) {
                int j = particleIndices[k];
                if (i == j || j > currActiveIndex) continue;

                // impulse based collision
                Vec2<float> delta = position[i] - position[j];
                if (delta.lengthSq() == 0
                    || delta.lengthSq() > powf(d_radius + d_radius, 2)) continue;
                Vec2<float> deltaNorm = delta.normalized();
                float overlap = (d_radius + d_radius) - delta.length();
                posDelta += deltaNorm * overlap / 2.5f;

                Vec2<float> relativeVelocity = velocity[i] - velocity[j];
                float dotProd = dot(relativeVelocity, deltaNorm);
                if (dotProd <= 0) {
                    float impulse = 2 * dotProd / (d_mass + d_mass);
                    impulse = clamp(impulse, -4.0f, 4.0f);
                    velDelta -= deltaNorm * impulse * d_mass * d_restitution;
                }

                // position based dynamics
                // doesn't work too well compared to impulse based collision tho
                // Vec2<float> delta = position[j] - position[i];
                // float distSqr = delta.lengthSq();
                // if (delta.lengthSq() == 0
                //     || distSqr > powf(d_radius + d_radius, 2)) continue;
                // float dist = sqrt(distSqr);
                // float overlap = 0.5f * (d_radius + d_radius - dist) / dist;
                // Vec2<float> displacement = delta * overlap;
                // posDelta.x = posDelta.x - displacement.x;
                // posDelta.y = posDelta.y - displacement.y;
            }
        }
    }

    // udpate the shared memory with each particle's delta from the collision
    atomicAdd(&posDeltaX, posDelta.x);
    atomicAdd(&posDeltaY, posDelta.y);
    atomicAdd(&velDeltaX, velDelta.x);
    atomicAdd(&velDeltaY, velDelta.y);
    __syncthreads();

    // only the first thread in the block will update the shared memory
    if (threadIdx.x != 0) return;

    // coalesce the shared memory into the final position and velocity output
    velocity[i] = velocity[i] + Vec2<float>(velDeltaX, velDeltaY);
    velocity[i] *= powf(d_dampingFactor, d_dampingFactorRate * deltaTime);
    velocity[i].y += gravity * deltaTime;
    position[i] = position[i] + Vec2<float>(posDeltaX, posDeltaY) + velocity[i] * deltaTime;

    // check for border collisions
    if (position[i].x - d_radius < d_borderLeft) {
        position[i].x = d_borderLeft + d_radius;
        velocity[i].x *= -1 * d_restitution;
    }
    if (position[i].x + d_radius > d_borderRight) {
        position[i].x = d_borderRight - d_radius;
        velocity[i].x *= -1 * d_restitution;
    }
    if (position[i].y - d_radius < d_borderTop) {
        position[i].y = d_borderTop + d_radius;
        velocity[i].y *= -1 * d_restitution;
    }
    if (position[i].y + d_radius > d_borderBottom) {
        position[i].y = d_borderBottom - d_radius;
        velocity[i].y *= -1 * d_restitution;
    }
}

void Particles::update(float deltaTime, float gravity) {
    hipStream_t stream;
    cudaAssert(hipStreamCreate(&stream));

    dim3 blocks = maxParticleCount;
    dim3 threads = min(maxParticleCount, gpu_maxThreadCount);

    if (spawn && (currActiveIndex == -1 || currActiveIndex < maxParticleCount)) {
        currActiveIndex = (currActiveIndex == -1) ? 0 : currActiveIndex;
        spawnParticleKernel<<<1, min(spawnCount, maxParticleCount - spawnCount), 0, stream>>>(d_position, d_velocity, 
            Vec2<float>(static_cast<float>(mouseXPos), static_cast<float>(mouseYPos)), currActiveIndex);
        currActiveIndex += min(spawnCount, maxParticleCount - spawnCount);
    }
    if (succ && currActiveIndex != -1) {
        succParticlesKernel<<<blocks, threads, 0, stream>>>(d_position, d_velocity, currActiveIndex, 
            Vec2<float>(static_cast<float>(mouseXPos), static_cast<float>(mouseYPos)));
    }
    if (repel && currActiveIndex != -1) {
        repelParticlesKernel<<<blocks, threads, 0, stream>>>(d_position, d_velocity, currActiveIndex, 
            Vec2<float>(static_cast<float>(mouseXPos), static_cast<float>(mouseYPos)));
    }  

    createSpatialHashTable<<<1, threads, 0, stream>>>(d_position, d_spatialHashTable, d_particleIndices, cellSize);
    cudaAssert(hipStreamSynchronize(stream));

    // this is the iterative solver loop for the position based dynamics, unused for now
    // for (int iter = 0; iter < 4; iter++) {
    updateKernel<<<blocks, threads, 0, stream>>>(d_position, d_velocity, currActiveIndex,
        d_spatialHashTable, d_particleIndices, deltaTime, gravity, cellSize);
    // positionBasedDynamicsKernel<<<1, 1, 0, stream>>>(d_position, d_velocity, currActiveIndex,
    //     d_spatialHashTable, d_particleIndices, deltaTime, gravity, cellSize);
    cudaAssert(hipStreamSynchronize(stream));
    // }

    cudaAssert(hipMemcpyAsync(position.data(), d_position, maxParticleCount * sizeof(Vec2<float>), hipMemcpyDeviceToHost, stream));
    cudaAssert(hipStreamDestroy(stream));
}
